#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <math.h>

// CUDA Error Checking Macro
#define CUDA_CHECK(call) do {     hipError_t err = call;     if (err != hipSuccess) {         fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__,                 hipGetErrorString(err));         exit(1);     } } while(0)


template<typename T>
struct Tensor {
  T* data;
  const int* shape;
  int dims;

  __device__ inline T& operator()(int i) { return data[i]; }
  __device__ inline T& operator()(int i, int j) { return data[i * shape[1] + j]; }
  __device__ inline T& operator()(int i, int j, int k) { return data[(i * shape[1] + j) * shape[2] + k]; }
};


// ======================================================
// Kernel Definitions
// ======================================================

      /**
       * @cuda global
       */
      __global__ void embedding_forward(Tensor<float> output, Tensor<int> input, Tensor<float> embeddings) {
        int batch_idx = blockIdx.y;
        int seq_idx = blockIdx.x;

        if (batch_idx < input.shape[0] && seq_idx < input.shape[1]) {
          int token_id = input(batch_idx, seq_idx);
          for (int i = threadIdx.x; i < output.shape[2]; i += blockDim.x) {
            output(batch_idx, seq_idx, i) = embeddings(token_id, i);
          }
        }
      }
    


      /**
       * @cuda global
       */
      __global__ void positional_encoding_forward(Tensor<float> output, Tensor<float> input) {
        int batch_idx = blockIdx.z;
        int seq_idx = blockIdx.y;
        int embed_idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (batch_idx < input.shape[0] && seq_idx < input.shape[1] && embed_idx < input.shape[2]) {
          float pos = (float)seq_idx;
          float i = (float)embed_idx;
          float val;
          if (embed_idx % 2 == 0) {
            val = sinf(pos / powf(10000.0f, (2.0f * i) / (float)input.shape[2]));
          } else {
            val = cosf(pos / powf(10000.0f, (2.0f * i) / (float)input.shape[2]));
          }
          output(batch_idx, seq_idx, embed_idx) = input(batch_idx, seq_idx, embed_idx) + val;
        }
      }
    


      /**
       * @cuda global
       */
      __global__ void dense_forward(
        Tensor<float> output, 
        Tensor<float> input, 
        Tensor<float> weights, 
        Tensor<float> bias
      ) {
        // Each thread computes one output element.
        // Grid: (output_features / threads_per_block, batch_size)
        // Block: (threads_per_block)
        int batch_idx = blockIdx.y;
        int output_feature_idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (batch_idx < input.shape[0] && output_feature_idx < output.shape[1]) {
          float sum = 0.0f;
          for (int k = 0; k < input.shape[1]; ++k) {
            sum += input(batch_idx, k) * weights(k, output_feature_idx);
          }
          output(batch_idx, output_feature_idx) = sum + bias(output_feature_idx);
        }
      }
    


      /**
       * @cuda global
       */
      __global__ void split_heads_forward(Tensor<float> output, Tensor<float> input) {
        // Input: [batch, seq_len, embed_dim]
        // Output: [batch, num_heads, seq_len, head_dim]
        int batch_idx = blockIdx.z;
        int seq_idx = blockIdx.y;
        int head_idx = blockIdx.x;
        int feature_idx = threadIdx.x;

        int num_heads = output.shape[1];
        int head_dim = output.shape[3];

        if (batch_idx < input.shape[0] && seq_idx < input.shape[1] && head_idx < num_heads && feature_idx < head_dim) {
          int embed_idx = head_idx * head_dim + feature_idx;
          output(batch_idx, head_idx, seq_idx, feature_idx) = input(batch_idx, seq_idx, embed_idx);
        }
      }
    


      /**
       * @cuda global
       */
      __global__ void batched_matmul_transpose_b(Tensor<float> output, Tensor<float> a, Tensor<float> b) {
        int batch_idx = blockIdx.z;
        int head_idx = blockIdx.y;
        int row = blockIdx.x;
        int col = threadIdx.x;

        if (batch_idx < a.shape[0] && head_idx < a.shape[1] && row < a.shape[2] && col < output.shape[3]) {
          float sum = 0.0f;
          for (int k = 0; k < a.shape[3]; ++k) {
            sum += a(batch_idx, head_idx, row, k) * b(batch_idx, head_idx, col, k);
          }
          output(batch_idx, head_idx, row, col) = sum;
        }
      }
    


      /**
       * @cuda global
       */
      __global__ void scale_forward(Tensor<float> output, Tensor<float> input) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int size = input.shape[0] * input.shape[1] * input.shape[2] * input.shape[3];
        
        for (int i = idx; i < size; i += gridDim.x * blockDim.x) {
            output.data[i] = input.data[i] * 0.17677669529663687;
        }
      }
    


      /**
       * @cuda global
       */
      __global__ void softmax_forward(Tensor<float> output, Tensor<float> input) {
        // This kernel computes softmax over the last dimension.
        // It handles both 2D tensors [batch, features] and 4D tensors [batch, heads, seq, seq]
        extern __shared__ float shared_mem[];
        int tid = threadIdx.x;
        
        if (input.dims == 2) {
          // Handle 2D case: [batch, features]
          int batch_idx = blockIdx.x;
          int size = input.shape[1];
          
          // 1. Find max for numerical stability
          float max_val = -FLT_MAX;
          for (int i = tid; i < size; i += blockDim.x) {
              max_val = fmaxf(max_val, input(batch_idx, i));
          }
          shared_mem[tid] = max_val;
          __syncthreads();
          for (int s = blockDim.x / 2; s > 0; s >>= 1) {
              if (tid < s) { shared_mem[tid] = fmaxf(shared_mem[tid], shared_mem[tid + s]); }
              __syncthreads();
          }
          max_val = shared_mem[0];

          // 2. Calculate sum of exps
          float sum_exp = 0.0f;
          for (int i = tid; i < size; i += blockDim.x) {
              sum_exp += expf(input(batch_idx, i) - max_val);
          }
          shared_mem[tid] = sum_exp;
          __syncthreads();
          for (int s = blockDim.x / 2; s > 0; s >>= 1) {
              if (tid < s) { shared_mem[tid] += shared_mem[tid + s]; }
              __syncthreads();
          }
          sum_exp = shared_mem[0];

          // 3. Calculate softmax
          for (int i = tid; i < size; i += blockDim.x) {
              output(batch_idx, i) = expf(input(batch_idx, i) - max_val) / sum_exp;
          }
        } else {
          // Handle 4D case: [batch, heads, seq, seq]
          int batch_idx = blockIdx.z;
          int head_idx = blockIdx.y;
          int row_idx = blockIdx.x;
          int size = input.shape[3];

          // 1. Find max for numerical stability
          float max_val = -FLT_MAX;
          for (int i = tid; i < size; i += blockDim.x) {
              max_val = fmaxf(max_val, input(batch_idx, head_idx, row_idx, i));
          }
          shared_mem[tid] = max_val;
          __syncthreads();
          for (int s = blockDim.x / 2; s > 0; s >>= 1) {
              if (tid < s) { shared_mem[tid] = fmaxf(shared_mem[tid], shared_mem[tid + s]); }
              __syncthreads();
          }
          max_val = shared_mem[0];

          // 2. Calculate sum of exps
          float sum_exp = 0.0f;
          for (int i = tid; i < size; i += blockDim.x) {
              sum_exp += expf(input(batch_idx, head_idx, row_idx, i) - max_val);
          }
          shared_mem[tid] = sum_exp;
          __syncthreads();
          for (int s = blockDim.x / 2; s > 0; s >>= 1) {
              if (tid < s) { shared_mem[tid] += shared_mem[tid + s]; }
              __syncthreads();
          }
          sum_exp = shared_mem[0];

          // 3. Calculate softmax
          for (int i = tid; i < size; i += blockDim.x) {
              output(batch_idx, head_idx, row_idx, i) = expf(input(batch_idx, head_idx, row_idx, i) - max_val) / sum_exp;
          }
        }
      }
    


      /**
       * @cuda global
       */
      __global__ void batched_matmul(Tensor<float> output, Tensor<float> a, Tensor<float> b) {
        int batch_idx = blockIdx.z;
        int head_idx = blockIdx.y;
        int row = blockIdx.x;
        int col = threadIdx.x;

        if (batch_idx < a.shape[0] && head_idx < a.shape[1] && row < a.shape[2] && col < output.shape[3]) {
          float sum = 0.0f;
          for (int k = 0; k < a.shape[3]; ++k) {
            sum += a(batch_idx, head_idx, row, k) * b(batch_idx, head_idx, k, col);
          }
          output(batch_idx, head_idx, row, col) = sum;
        }
      }
    


      /**
       * @cuda global
       */
      __global__ void concat_heads_forward(Tensor<float> output, Tensor<float> input) {
        // Input: [batch, num_heads, seq_len, head_dim]
        // Output: [batch, seq_len, embed_dim]
        int batch_idx = blockIdx.z;
        int seq_idx = blockIdx.y;
        int head_idx = blockIdx.x;
        int feature_idx = threadIdx.x;

        int num_heads = input.shape[1];
        int head_dim = input.shape[3];

        if (batch_idx < output.shape[0] && seq_idx < output.shape[1] && head_idx < num_heads && feature_idx < head_dim) {
          int embed_idx = head_idx * head_dim + feature_idx;
          output(batch_idx, seq_idx, embed_idx) = input(batch_idx, head_idx, seq_idx, feature_idx);
        }
      }
    


      /**
       * @cuda global
       */
      __global__ void add_forward(Tensor<float> output, Tensor<float> a, Tensor<float> b) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        // This is a simplified approach. A robust implementation would handle
        // arbitrary dimensions and calculate total size on the host.
        int size = a.shape[0] * a.shape[1] * a.shape[2];

        for (int i = idx; i < size; i += gridDim.x * blockDim.x) {
            // This assumes flattened indexing. A better way is to reconstruct multidim index.
            // For now, we'll assume the shapes are identical and can be treated as flat arrays.
            output.data[i] = a.data[i] + b.data[i];
        }
      }
    


      /**
       * @cuda global
       */
      __global__ void layer_norm_forward(
        Tensor<float> output,
        Tensor<float> input,
        Tensor<float> gamma,
        Tensor<float> beta
      ) {
        // This kernel processes one feature vector (e.g., one token's embedding) per block.
        // Grid: (batch_size, seq_len)
        // Block: (feature_count)
        extern __shared__ float shared_mem[];
        int batch_idx = blockIdx.y;
        int seq_idx = blockIdx.x;
        int feature_count = input.shape[2];
        int tid = threadIdx.x;

        // Step 1: Calculate mean
        float sum = 0.0f;
        for (int i = tid; i < feature_count; i += blockDim.x) {
            sum += input(batch_idx, seq_idx, i);
        }
        shared_mem[tid] = sum;
        __syncthreads();

        // Parallel reduction for mean
        for (int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (tid < s) {
                shared_mem[tid] += shared_mem[tid + s];
            }
            __syncthreads();
        }
        float mean = shared_mem[0] / feature_count;

        // Step 2: Calculate variance
        sum = 0.0f;
        for (int i = tid; i < feature_count; i += blockDim.x) {
            float dev = input(batch_idx, seq_idx, i) - mean;
            sum += dev * dev;
        }
        shared_mem[tid] = sum;
        __syncthreads();
        
        // Parallel reduction for variance
        for (int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (tid < s) {
                shared_mem[tid] += shared_mem[tid + s];
            }
            __syncthreads();
        }
        float variance = shared_mem[0] / feature_count;
        float rsqrt_variance = rsqrtf(variance + 0.00001);

        // Step 3: Normalize
        for (int i = tid; i < feature_count; i += blockDim.x) {
            float normalized = (input(batch_idx, seq_idx, i) - mean) * rsqrt_variance;
            output(batch_idx, seq_idx, i) = normalized * gamma(i) + beta(i);
        }
      }
    


      /**
       * @cuda global
       */
      __global__ void relu_forward(Tensor<float> output, Tensor<float> input) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int size = 1;
        for (int i = 0; i < input.dims; ++i) {
          size *= input.shape[i];
        }
        if (idx < size) {
          output(idx) = fmaxf(0.0f, input(idx));
        }
      }
    

// ======================================================
// Main Host-Side Execution Function
// ======================================================
extern "C" void executeGraph(
  float* input_data,
  const int* input_shape,
  int input_dims,
  float* output_data,
  const int* output_shape,
  int output_dims,
  float* embeddings_data,
  const int* embeddings_shape,
  int embeddings_dims,
  float* weights_data,
  const int* weights_shape,
  int weights_dims,
  float* bias_data,
  const int* bias_shape,
  int bias_dims,
  float* gamma_data,
  const int* gamma_shape,
  int gamma_dims,
  float* beta_data,
  const int* beta_shape,
  int beta_dims,
  char* workspace
) {
  // --- Variable Declarations ---
  const int intermediate_0_shape[] = {1, 256, 128};
  const int intermediate_1_shape[] = {1, 256, 128};
  const int intermediate_2_shape[] = {1, 128};
  const int intermediate_3_shape[] = {1, 128};
  const int intermediate_4_shape[] = {1, 128};
  const int intermediate_5_shape[] = {1, 4, 128, 32};
  const int intermediate_6_shape[] = {1, 4, 128, 32};
  const int intermediate_7_shape[] = {1, 4, 128, 32};
  const int intermediate_8_shape[] = {1, 4, 128, 128};
  const int intermediate_9_shape[] = {1, 4, 128, 128};
  const int intermediate_10_shape[] = {1, 4, 128, 128};
  const int intermediate_11_shape[] = {1, 4, 128, 32};
  const int intermediate_12_shape[] = {1, 128, 128};
  const int intermediate_13_shape[] = {1, 128};
  const int intermediate_14_shape[] = {1, 256, 128};
  const int intermediate_15_shape[] = {1, 256, 128};
  const int intermediate_16_shape[] = {1, 512};
  const int intermediate_17_shape[] = {1, 512};
  const int intermediate_18_shape[] = {1, 128};
  const int intermediate_19_shape[] = {1, 256, 128};
  const int intermediate_20_shape[] = {1, 256, 128};
  const int intermediate_21_shape[] = {1, 128};
  const int intermediate_22_shape[] = {1, 128};
  const int intermediate_23_shape[] = {1, 128};
  const int intermediate_24_shape[] = {1, 4, 128, 32};
  const int intermediate_25_shape[] = {1, 4, 128, 32};
  const int intermediate_26_shape[] = {1, 4, 128, 32};
  const int intermediate_27_shape[] = {1, 4, 128, 128};
  const int intermediate_28_shape[] = {1, 4, 128, 128};
  const int intermediate_29_shape[] = {1, 4, 128, 128};
  const int intermediate_30_shape[] = {1, 4, 128, 32};
  const int intermediate_31_shape[] = {1, 128, 128};
  const int intermediate_32_shape[] = {1, 128};
  const int intermediate_33_shape[] = {1, 256, 128};
  const int intermediate_34_shape[] = {1, 256, 128};
  const int intermediate_35_shape[] = {1, 512};
  const int intermediate_36_shape[] = {1, 512};
  const int intermediate_37_shape[] = {1, 128};
  const int intermediate_38_shape[] = {1, 256, 128};
  const int intermediate_39_shape[] = {1, 256, 128};
  const int intermediate_40_shape[] = {1, 1000};

  // --- Tensor Struct Instantiation ---
  Tensor<float> intermediate_0_tensor = {(float*)(workspace + 0), intermediate_0_shape, 3};
  Tensor<float> intermediate_1_tensor = {(float*)(workspace + 131072), intermediate_1_shape, 3};
  Tensor<float> intermediate_2_tensor = {(float*)(workspace + 0), intermediate_2_shape, 2};
  Tensor<float> intermediate_3_tensor = {(float*)(workspace + 262144), intermediate_3_shape, 2};
  Tensor<float> intermediate_4_tensor = {(float*)(workspace + 262656), intermediate_4_shape, 2};
  Tensor<float> intermediate_5_tensor = {(float*)(workspace + 263168), intermediate_5_shape, 4};
  Tensor<float> intermediate_6_tensor = {(float*)(workspace + 0), intermediate_6_shape, 4};
  Tensor<float> intermediate_7_tensor = {(float*)(workspace + 328704), intermediate_7_shape, 4};
  Tensor<float> intermediate_8_tensor = {(float*)(workspace + 394240), intermediate_8_shape, 4};
  Tensor<float> intermediate_9_tensor = {(float*)(workspace + 656384), intermediate_9_shape, 4};
  Tensor<float> intermediate_10_tensor = {(float*)(workspace + 394240), intermediate_10_shape, 4};
  Tensor<float> intermediate_11_tensor = {(float*)(workspace + 0), intermediate_11_shape, 4};
  Tensor<float> intermediate_12_tensor = {(float*)(workspace + 263168), intermediate_12_shape, 3};
  Tensor<float> intermediate_13_tensor = {(float*)(workspace + 0), intermediate_13_shape, 2};
  Tensor<float> intermediate_14_tensor = {(float*)(workspace + 394240), intermediate_14_shape, 3};
  Tensor<float> intermediate_15_tensor = {(float*)(workspace + 0), intermediate_15_shape, 3};
  Tensor<float> intermediate_16_tensor = {(float*)(workspace + 131072), intermediate_16_shape, 2};
  Tensor<float> intermediate_17_tensor = {(float*)(workspace + 263168), intermediate_17_shape, 2};
  Tensor<float> intermediate_18_tensor = {(float*)(workspace + 131072), intermediate_18_shape, 2};
  Tensor<float> intermediate_19_tensor = {(float*)(workspace + 394240), intermediate_19_shape, 3};
  Tensor<float> intermediate_20_tensor = {(float*)(workspace + 0), intermediate_20_shape, 3};
  Tensor<float> intermediate_21_tensor = {(float*)(workspace + 131072), intermediate_21_shape, 2};
  Tensor<float> intermediate_22_tensor = {(float*)(workspace + 262144), intermediate_22_shape, 2};
  Tensor<float> intermediate_23_tensor = {(float*)(workspace + 262656), intermediate_23_shape, 2};
  Tensor<float> intermediate_24_tensor = {(float*)(workspace + 263168), intermediate_24_shape, 4};
  Tensor<float> intermediate_25_tensor = {(float*)(workspace + 131072), intermediate_25_shape, 4};
  Tensor<float> intermediate_26_tensor = {(float*)(workspace + 328704), intermediate_26_shape, 4};
  Tensor<float> intermediate_27_tensor = {(float*)(workspace + 394240), intermediate_27_shape, 4};
  Tensor<float> intermediate_28_tensor = {(float*)(workspace + 656384), intermediate_28_shape, 4};
  Tensor<float> intermediate_29_tensor = {(float*)(workspace + 394240), intermediate_29_shape, 4};
  Tensor<float> intermediate_30_tensor = {(float*)(workspace + 131072), intermediate_30_shape, 4};
  Tensor<float> intermediate_31_tensor = {(float*)(workspace + 263168), intermediate_31_shape, 3};
  Tensor<float> intermediate_32_tensor = {(float*)(workspace + 131072), intermediate_32_shape, 2};
  Tensor<float> intermediate_33_tensor = {(float*)(workspace + 394240), intermediate_33_shape, 3};
  Tensor<float> intermediate_34_tensor = {(float*)(workspace + 0), intermediate_34_shape, 3};
  Tensor<float> intermediate_35_tensor = {(float*)(workspace + 131072), intermediate_35_shape, 2};
  Tensor<float> intermediate_36_tensor = {(float*)(workspace + 263168), intermediate_36_shape, 2};
  Tensor<float> intermediate_37_tensor = {(float*)(workspace + 131072), intermediate_37_shape, 2};
  Tensor<float> intermediate_38_tensor = {(float*)(workspace + 394240), intermediate_38_shape, 3};
  Tensor<float> intermediate_39_tensor = {(float*)(workspace + 0), intermediate_39_shape, 3};
  Tensor<float> intermediate_40_tensor = {(float*)(workspace + 131072), intermediate_40_shape, 2};
  Tensor<float> input = {input_data, input_shape, input_dims};
  Tensor<float> output = {output_data, output_shape, output_dims};
  Tensor<float> embeddings = {embeddings_data, embeddings_shape, embeddings_dims};
  Tensor<float> weights = {weights_data, weights_shape, weights_dims};
  Tensor<float> bias = {bias_data, bias_shape, bias_dims};
  Tensor<float> gamma = {gamma_data, gamma_shape, gamma_dims};
  Tensor<float> beta = {beta_data, beta_shape, beta_dims};

  // --- Kernel Launch Sequence ---
  embedding_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_0_tensor, input, embeddings);
  CUDA_CHECK(hipGetLastError());
  positional_encoding_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_1_tensor, intermediate_0_tensor);
  CUDA_CHECK(hipGetLastError());
  dense_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_2_tensor, intermediate_1_tensor, weights, bias);
  CUDA_CHECK(hipGetLastError());
  dense_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_3_tensor, intermediate_1_tensor, weights, bias);
  CUDA_CHECK(hipGetLastError());
  dense_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_4_tensor, intermediate_1_tensor, weights, bias);
  CUDA_CHECK(hipGetLastError());
  split_heads_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_5_tensor, intermediate_2_tensor);
  CUDA_CHECK(hipGetLastError());
  split_heads_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_6_tensor, intermediate_3_tensor);
  CUDA_CHECK(hipGetLastError());
  split_heads_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_7_tensor, intermediate_4_tensor);
  CUDA_CHECK(hipGetLastError());
  batched_matmul_transpose_b<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_8_tensor, intermediate_5_tensor, intermediate_6_tensor);
  CUDA_CHECK(hipGetLastError());
  scale_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_9_tensor, intermediate_8_tensor);
  CUDA_CHECK(hipGetLastError());
  softmax_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 1024>>>(intermediate_10_tensor, intermediate_9_tensor);
  CUDA_CHECK(hipGetLastError());
  batched_matmul<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_11_tensor, intermediate_10_tensor, intermediate_7_tensor);
  CUDA_CHECK(hipGetLastError());
  concat_heads_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_12_tensor, intermediate_11_tensor);
  CUDA_CHECK(hipGetLastError());
  dense_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_13_tensor, intermediate_12_tensor, weights, bias);
  CUDA_CHECK(hipGetLastError());
  add_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_14_tensor, intermediate_1_tensor, intermediate_13_tensor);
  CUDA_CHECK(hipGetLastError());
  layer_norm_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 1024>>>(intermediate_15_tensor, intermediate_14_tensor, gamma, beta);
  CUDA_CHECK(hipGetLastError());
  dense_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_16_tensor, intermediate_15_tensor, weights, bias);
  CUDA_CHECK(hipGetLastError());
  relu_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_17_tensor, intermediate_16_tensor);
  CUDA_CHECK(hipGetLastError());
  dense_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_18_tensor, intermediate_17_tensor, weights, bias);
  CUDA_CHECK(hipGetLastError());
  add_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_19_tensor, intermediate_15_tensor, intermediate_18_tensor);
  CUDA_CHECK(hipGetLastError());
  layer_norm_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 1024>>>(intermediate_20_tensor, intermediate_19_tensor, gamma, beta);
  CUDA_CHECK(hipGetLastError());
  dense_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_21_tensor, intermediate_20_tensor, weights, bias);
  CUDA_CHECK(hipGetLastError());
  dense_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_22_tensor, intermediate_20_tensor, weights, bias);
  CUDA_CHECK(hipGetLastError());
  dense_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_23_tensor, intermediate_20_tensor, weights, bias);
  CUDA_CHECK(hipGetLastError());
  split_heads_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_24_tensor, intermediate_21_tensor);
  CUDA_CHECK(hipGetLastError());
  split_heads_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_25_tensor, intermediate_22_tensor);
  CUDA_CHECK(hipGetLastError());
  split_heads_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_26_tensor, intermediate_23_tensor);
  CUDA_CHECK(hipGetLastError());
  batched_matmul_transpose_b<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_27_tensor, intermediate_24_tensor, intermediate_25_tensor);
  CUDA_CHECK(hipGetLastError());
  scale_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_28_tensor, intermediate_27_tensor);
  CUDA_CHECK(hipGetLastError());
  softmax_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 1024>>>(intermediate_29_tensor, intermediate_28_tensor);
  CUDA_CHECK(hipGetLastError());
  batched_matmul<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_30_tensor, intermediate_29_tensor, intermediate_26_tensor);
  CUDA_CHECK(hipGetLastError());
  concat_heads_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_31_tensor, intermediate_30_tensor);
  CUDA_CHECK(hipGetLastError());
  dense_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_32_tensor, intermediate_31_tensor, weights, bias);
  CUDA_CHECK(hipGetLastError());
  add_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_33_tensor, intermediate_20_tensor, intermediate_32_tensor);
  CUDA_CHECK(hipGetLastError());
  layer_norm_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 1024>>>(intermediate_34_tensor, intermediate_33_tensor, gamma, beta);
  CUDA_CHECK(hipGetLastError());
  dense_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_35_tensor, intermediate_34_tensor, weights, bias);
  CUDA_CHECK(hipGetLastError());
  relu_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_36_tensor, intermediate_35_tensor);
  CUDA_CHECK(hipGetLastError());
  dense_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_37_tensor, intermediate_36_tensor, weights, bias);
  CUDA_CHECK(hipGetLastError());
  add_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_38_tensor, intermediate_34_tensor, intermediate_37_tensor);
  CUDA_CHECK(hipGetLastError());
  layer_norm_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 1024>>>(intermediate_39_tensor, intermediate_38_tensor, gamma, beta);
  CUDA_CHECK(hipGetLastError());
  dense_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 0>>>(intermediate_40_tensor, intermediate_39_tensor, weights, bias);
  CUDA_CHECK(hipGetLastError());
  softmax_forward<<<dim3(1, 1, 1), dim3(256, 1, 1), 1024>>>(output, intermediate_40_tensor);
  CUDA_CHECK(hipGetLastError());
  // --- End Execution Flow ---
}
    