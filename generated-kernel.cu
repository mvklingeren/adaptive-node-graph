
#include <hip/hip_runtime.h>
#include <math.h>


template<typename T>
struct Tensor {
  T* data;
  const int* shape;
  int dims;

  __device__ inline T& operator()(int i) {
    return data[i];
  }

  __device__ inline T& operator()(int i, int j) {
    return data[i * shape[1] + j];
  }

  __device__ inline T& operator()(int i, int j, int k) {
    return data[(i * shape[1] + j) * shape[2] + k];
  }
};


// ======================================================
// Node Device Functions
// ======================================================

      __device__ void dense_forward_0(
        Tensor<float> output, 
        Tensor<float> input, 
        Tensor<float> weights_0, 
        Tensor<float> bias_0
      ) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        
        int input_size = input.shape[0];
        int output_size = output.shape[0];

        if (i < output_size) {
          float sum = 0.0f;
          for (int j = 0; j < input_size; ++j) {
            sum += input(j) * weights_0(i, j);
          }
          output(i) = sum + bias_0(i);
        }
      }
    


      __device__ void relu_forward(Tensor<float> output, Tensor<float> input) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        int size = 1;
        for (int i = 0; i < input.dims; ++i) {
          size *= input.shape[i];
        }
        if (idx < size) {
          output(idx) = fmaxf(0.0f, input(idx));
        }
      }
    


      __device__ void dense_forward_1(
        Tensor<float> output, 
        Tensor<float> input, 
        Tensor<float> weights_1, 
        Tensor<float> bias_1
      ) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        
        int input_size = input.shape[0];
        int output_size = output.shape[0];

        if (i < output_size) {
          float sum = 0.0f;
          for (int j = 0; j < input_size; ++j) {
            sum += input(j) * weights_1(i, j);
          }
          output(i) = sum + bias_1(i);
        }
      }
    

// ======================================================
// Main Fused Graph Kernel
// ======================================================
extern "C" __global__ void executeGraph(
  float* input_data,
  const int* input_shape,
  int input_dims,
  float* output_data,
  const int* output_shape,
  int output_dims,
  float* weights_0_data,
  const int* weights_0_shape,
  int weights_0_dims,
  float* bias_0_data,
  const int* bias_0_shape,
  int bias_0_dims,
  float* weights_1_data,
  const int* weights_1_shape,
  int weights_1_dims,
  float* bias_1_data,
  const int* bias_1_shape,
  int bias_1_dims,
  char* workspace
) {
  // TODO: Implement tensor-based indexing
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // --- Variable Declarations ---
  const int intermediate_0_shape[] = {256};
  const int intermediate_1_shape[] = {256};

  // --- Tensor Struct Instantiation ---
  Tensor<float> intermediate_0_tensor = {(float*)(workspace + 0), intermediate_0_shape, 1};
  Tensor<float> intermediate_1_tensor = {(float*)(workspace + 0), intermediate_1_shape, 1};
  Tensor<float> input = {input_data, input_shape, input_dims};
  Tensor<float> output = {output_data, output_shape, output_dims};
  Tensor<float> weights_0 = {weights_0_data, weights_0_shape, weights_0_dims};
  Tensor<float> bias_0 = {bias_0_data, bias_0_shape, bias_0_dims};
  Tensor<float> weights_1 = {weights_1_data, weights_1_shape, weights_1_dims};
  Tensor<float> bias_1 = {bias_1_data, bias_1_shape, bias_1_dims};

  // --- Generated Execution Flow ---
    dense_forward_0(intermediate_0_tensor, input, weights_0, bias_0);
    relu_forward(intermediate_1_tensor, intermediate_0_tensor);
    dense_forward_1(output, intermediate_1_tensor, weights_1, bias_1);
  // --- End Execution Flow ---
}
    